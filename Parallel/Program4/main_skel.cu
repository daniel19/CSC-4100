
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <stdlib.h>
#include <iomanip>
void  printGrid(float* grid, int rows, int cols);
/**
 *Kernel will update the matrix to keep the heater cells constant.
 */
__global__ void copyHeaters(float* stateGrid, float* heaterGrid, int nRows, int nCols, int iteration) {
    dim3 gIdx;
    gIdx.y = blockIdx.y * blockDim.y + threadIdx.y; //row
    gIdx.x = blockIdx.x * blockDim.x + threadIdx.x; // col
    int i = gIdx.y;
    int j = gIdx.x;
   
    if(gIdx.x < nCols && gIdx.y < nRows){
            float heatValue = heaterGrid[i*nCols +j];
            if(heatValue != 0)
                stateGrid[i*nCols + j] =heatValue;
    }
}

__global__ void updateGrid(float* inGrid, float* outGrid, float k, int nRows, int nCols) {
    dim3 gIdx;
    gIdx.y = blockIdx.y * blockDim.y + threadIdx.y; //row
    gIdx.x = blockIdx.x * blockDim.x + threadIdx.x; // col
    int i = gIdx.y;
    int j = gIdx.x;

    //Find these values from the inGrid
    int Tlft, Trite, Tup, Tdown;
    if(gIdx.x < nCols && gIdx.y < nRows){
           int currentPosition = i*nCols+j;
           Tlft = currentPosition -1;
           Trite = currentPosition +1;
           Tup = currentPosition -nCols;
           Tdown = currentPosition +nCols;
           float Tnew = inGrid[currentPosition];
           float Top, Tbottom, Tleft, Tright;
           Tbottom = (Tdown >= nCols*nRows) ? Tnew : inGrid[Tdown];
           Top = (Tup < 0) ? Tnew : inGrid[Tup];
           Tright = (Trite >= nCols*nRows) ?  Tnew : inGrid[Trite];
           Tleft = (Tlft < 0) ? Tnew : inGrid[Tlft]; 
           
           Tnew = Tnew + k*(Top + Tbottom + Tleft + Tright - (4*Tnew));

           outGrid[currentPosition] = Tnew;
   }
}

/*------------------------------------------------------------------------------
readHeaterFile

Assumes heaterGrid points to a flattened 2d array of size [rows,cols]
Fille heaterGrid with heaters from the heater file
------------------------------------------------------------------------------*/
void readHeaterFile(const char* fileName, float* heaterGrid, int rows, int cols) {
    std::ifstream inFile(fileName); 

    int numHeaters;
    inFile >> numHeaters;

    for(int i = 0; i < numHeaters; ++i) {
        int hRow, hCol;
        inFile >> hRow;
        inFile >> hCol;
        
        float temp;
        inFile >> temp; 

        heaterGrid[hRow * cols + hCol] = temp;
    }

    inFile.close();
}


/*------------------------------------------------------------------------------
printGrid
------------------------------------------------------------------------------*/
__device__ void printGrid(float* grid, int rows, int cols) {
    //std::cout << std::fixed << std::setprecision(2);
    for(int i = 0; i < rows; ++i) {
        for(int j = 0; j < cols; ++j) {
     //       std::cout << std::setw(6) << grid[i*cols+j] << " ";
            printf("%f ", grid[i*cols+j]);
        }
        //std::cout << std::endl;
        printf("\n");
    }
}

/*------------------------------------------------------------------------------
printGridToFile
------------------------------------------------------------------------------*/
void printGridToFile(float* grid, int rows, int cols, char* fileName) {
    std::ofstream outFile(fileName);
    outFile << std::fixed << std::setprecision(2);
    for(int i = 0; i < rows; ++i) {
        for(int j = 0; j < cols; ++j) {
            outFile << std::setw(6) << grid[i*cols+j] << " ";
        }
        outFile<< std::endl;
    }
    outFile.close();
}

/*------------------------------------------------------------------------------
main
------------------------------------------------------------------------------*/
int main(int argc, char** argv) {
    if(argc != 6) {
        std::cout << "Usage: " << argv[0] << 
            " <numRows> <numCols> <k> <timesteps> <heaterFileName>" << std::endl;

        return 0;
    }

    //Input arguments
    int rows = atoi(argv[1]);
    int cols = atoi(argv[2]);
    float k = atof(argv[3]);
    int timeSteps = atoi(argv[4]);

    //Allocate heater grid
    int gridSize = rows * cols * sizeof(float);
    float* heaterGrid_h = (float*)malloc(gridSize);

    //Read in heater file
    readHeaterFile(argv[5], heaterGrid_h, rows, cols); 

    float* heaterGrid_d; //device pointer
    //TODO Copy heater grid to device
    hipMalloc(&heaterGrid_d, gridSize);
    hipMemcpy(heaterGrid_d, heaterGrid_h, gridSize, hipMemcpyHostToDevice);


    //Input grid
    float* inGrid_h = (float*)malloc(gridSize);
    memset(inGrid_h, 0, gridSize);
    float* inGrid_d; //device pointer
    //TODO Allocate and copy inGrid to device
    hipMalloc(&inGrid_d, gridSize);
    hipMemcpy(inGrid_d, inGrid_h, gridSize, hipMemcpyHostToDevice);

    //Output grid
    float* outGrid_h = (float*)malloc(gridSize);
    memset(outGrid_h, 0, gridSize);
    float* outGrid_d; //device pointer
    //TODO Allocate and copy outGrid to device
    hipMalloc(&outGrid_d, gridSize);
    hipMemcpy(outGrid_d, outGrid_h, gridSize, hipMemcpyHostToDevice);

    dim3 bDim(16, 16);
    dim3 gDim;
    gDim.x = (rows + 16 - 1) / 16; //ceil(num_rows/16)
    gDim.y = (rows + 16 - 1) / 16;


    //TODO fill in update loop
    for(int i = 0; i < timeSteps; ++i) {
        //copy heater temps to inGrid_d (kernel call)
        copyHeaters<<<gDim, bDim>>>(inGrid_d, heaterGrid_d, rows, cols, i);

        //update outGrid_d based on inGrid_d (kernel call)
        updateGrid<<<gDim, bDim>>>(inGrid_d, outGrid_d, k, rows, cols);

        //swap pointers inGrid_d and outGrid_d
        float* temp = inGrid_d;
        inGrid_d = outGrid_d;
        outGrid_d = temp;
    }
    
    //TODO copy inGrid_d back to host (to inGrid_h)
    hipMemcpy(inGrid_h, inGrid_d, gridSize, hipMemcpyDeviceToHost); 
    printGridToFile(inGrid_h, rows, cols, "output_two.txt");

    return 0;
}
